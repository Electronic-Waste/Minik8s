#include <iostream>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include "hip/hip_runtime.h"

#include "hipblas.h"

#define M 2 // 
#define N 1 // 

#define BLOCK_SIZE 32  // 每个Block的线程数

void initial_array(float *array, int size)
{
    for(int i=0; i<size; i++)
    {
        array[i] = (float)(i);
    }
}

void print_array(float *array, int rows, int cols)
{
    for(int i=0; i<rows; i++)
    {
        for(int j=0; j<cols; j++)
        {
            std::cout << array[i*cols+j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

__global__ void matrix_product(float a[M][N],float b[N][M],float c[M][M])
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; 

    int j = threadIdx.y + blockIdx.y * blockDim.y; 

    if (i < M && j < M) 

    { 
        float tmp = 0;
        for (int m = 0;m < N;m++) {
            tmp += a[i][m]*b[m][j];
        }
        c[i][j] = tmp; 
    } 
}

void wrapper_product() 
{
    hipError_t cudaStat;
    // 申请内存
    int Axy = M * N;
    int Bxy = N * M;
    int Cxy = M * M;
    float *h_A, *h_B, *h_C;
    h_A = (float*)malloc(Axy * sizeof(float));
    h_B = (float*)malloc(Bxy * sizeof(float));
    h_C = (float*)malloc(Cxy * sizeof(float));

    // 初始化数组
    initial_array(h_A, Axy);
    initial_array(h_B, Bxy);

    // 申请显存
    float (*d_A)[N];
    float (*d_B)[M];
    float (*d_C)[M];
    cudaStat = hipMalloc((void**)&d_A, Axy * sizeof(float));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return;
    }
    cudaStat = hipMalloc((void**)&d_B, Bxy * sizeof(float));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return;
    }
    cudaStat = hipMalloc((void**)&d_C, Cxy * sizeof(float));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return;
    }
    hipMemcpy(d_A, h_A, Axy * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, Bxy * sizeof(float), hipMemcpyHostToDevice);

    dim3 DimGrid(1, 1); 

    dim3 DimBlock(M, M);

    matrix_product <<<DimGrid, DimBlock>>>(d_A, d_B, d_C);
    hipMemcpy(h_C,d_C,sizeof(float)*M*M,hipMemcpyDeviceToHost);
    std::cout << "finish matrix producting" << std::endl;
    print_array(h_C,M,M); 
}

int main() {
    wrapper_product();
    return 0;
}