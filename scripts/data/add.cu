#include <iostream>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include "hip/hip_runtime.h"

#include "hipblas.h"

#define M 32 // 
#define N 32 // 

#define BLOCK_SIZE 32  // 每个Block的线程数

void initial_array(float *array, int size)
{
    for(int i=0; i<size; i++)
    {
        array[i] = (float)(i);
    }
}

void print_array(float *array, int rows, int cols)
{
    for(int i=0; i<rows; i++)
    {
        for(int j=0; j<cols; j++)
        {
            std::cout << array[i*cols+j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

__global__ void matrix_add(float a[M][N],float b[M][N],float c[M][N])
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; 

    int j = threadIdx.y + blockIdx.y * blockDim.y; 

    if (i < M && j < N) 

    { 
        c[i][j] = a[i][j] + b[i][j]; 
    } 
}

void wrapper_add() 
{
    hipError_t cudaStat;
    // 申请内存
    int Axy = M * N;
    int Bxy = M * N;
    int Cxy = M * N;
    float *h_A, *h_B, *h_C;
    h_A = (float*)malloc(Axy * sizeof(float));
    h_B = (float*)malloc(Bxy * sizeof(float));
    h_C = (float*)malloc(Cxy * sizeof(float));

    // 初始化数组
    initial_array(h_A, Axy);
    initial_array(h_B, Bxy);

    // 申请显存
    float (*d_A)[N];
    float (*d_B)[N];
    float (*d_C)[N];
    cudaStat = hipMalloc((void**)&d_A, Axy * sizeof(float));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return;
    }
    cudaStat = hipMalloc((void**)&d_B, Bxy * sizeof(float));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return;
    }
    cudaStat = hipMalloc((void**)&d_C, Cxy * sizeof(float));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return;
    }
    hipMemcpy(d_A, h_A, Axy * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, Bxy * sizeof(float), hipMemcpyHostToDevice);

    dim3 DimGrid(1, 1); 

    dim3 DimBlock(32, 32);

    matrix_add <<<DimGrid, DimBlock>>>(d_A, d_B, d_C);
    hipMemcpy(h_C,d_C,sizeof(float)*M*N,hipMemcpyDeviceToHost);
    std::cout << "finish matrix adding" << std::endl;
    print_array(h_C,M,N); 
}

int main() {
    wrapper_add();
    return 0;
}