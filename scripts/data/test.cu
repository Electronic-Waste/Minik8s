#include <iostream>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include "hip/hip_runtime.h"

#include "hipblas.h"

#define M 8 // 矩阵行
#define K 8 // 矩阵列、矩阵行
#define N 8 // 矩阵列

#define BLOCK_SIZE 32  // 每个Block的线程数

// 初始化数组
void initial_array(float *array, int size)
{
    for(int i=0; i<size; i++)
    {
        array[i] = (float)(rand()%10+1);
    }
}

// 打印数组
void print_array(float *array, int rows, int cols)
{
    for(int i=0; i<rows; i++)
    {
        for(int j=0; j<cols; j++)
        {
            std::cout << array[i*cols+j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

void matrix_multiplication_cublas(int dimx_t, int dimy_t)
{
    hipError_t cudaStat;
    // 申请内存
    int Axy = M * K;
    int Bxy = K * N;
    int Cxy = M * N;
    float *h_A, *h_B, *h_C;
    h_A = (float*)malloc(Axy * sizeof(float));
    h_B = (float*)malloc(Bxy * sizeof(float));
    h_C = (float*)malloc(Cxy * sizeof(float));

    // 初始化数组
    initial_array(h_A, Axy);
    initial_array(h_B, Bxy);

    // 申请显存
    float *d_A, *d_B, *d_C;
    cudaStat = hipMalloc((void**)&d_A, Axy * sizeof(float));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return;
    }
    cudaStat = hipMalloc((void**)&d_B, Bxy * sizeof(float));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return;
    }
    cudaStat = hipMalloc((void**)&d_C, Cxy * sizeof(float));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return;
    }
    hipMemcpy(d_A, h_A, Axy * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, Bxy * sizeof(float), hipMemcpyHostToDevice);

    // 设置参数
    int dimx = dimx_t;
    int dimy = dimy_t;
    dim3 block(dimx, dimy);
    dim3 grid((M+block.x-1)/block.x, (N+block.y-1)/block.y);

    // 设置参数
    hipEvent_t gpustart, gpustop;
    float elapsedTime = 0.0;

    // 创建句柄
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    elapsedTime = 0.0;
    hipEventCreate(&gpustart);
    hipEventCreate(&gpustop);
    hipEventRecord(gpustart, 0);

    // 二维矩阵乘法-CUBLAS计算
    float a = 1, b = 0;
    hipblasSgemm(
            handle,
            HIPBLAS_OP_T,   // 矩阵A的属性参数，转置，按行优先
            HIPBLAS_OP_T,   // 矩阵B的属性参数，转置，按行优先
            M,             // 矩阵A行数、矩阵C行数
            N,             // 矩阵B列数、矩阵C列数
            K,             // 矩阵A列数、矩阵B行数
            &a,            // alpha的值
            d_A,           // 左矩阵，为A
            K,             // A的leading dimension，此时选择转置，按行优先，则leading dimension为A的列数
            d_B,           // 右矩阵，为B
            N,             // B的leading dimension，此时选择转置，按行优先，则leading dimension为B的列数
            &b,            // beta的值
            d_C,           // 结果矩阵C
            M              // C的leading dimension，C矩阵一定按列优先，则leading dimension为C的行数
    );
    hipMemcpy(h_C, d_C, Cxy * sizeof(float), hipMemcpyDeviceToHost); // 显存拷贝到内存
    hipDeviceSynchronize();
    hipEventRecord(gpustop, 0); // 记录结束时间
    hipEventSynchronize(gpustop);
    hipEventElapsedTime(&elapsedTime, gpustart, gpustop); // 计算耗时
    hipEventDestroy(gpustart);
    hipEventDestroy(gpustop);

    // 打印计算结果
    std::cout << "Matrix_A: " << M << "x" << K << std::endl;
    print_array(h_A, M, K);
    std::cout << "Matrix_B: " << K << "x" << N << std::endl;
    print_array(h_B, K, N);
    std::cout << "Matrix_C: " << M << "x" << N << std::endl;
    print_array(h_C, M, N);

    // 打印耗时
    printf("matrix_multiplication_cublas: ");
    printf("gridx: %4d, gridy: %4d, blockx: %4d, blocky: %4d", grid.x, grid.y, block.x, block.y);
    printf(", runtime: %8fs\n", elapsedTime/1000);

    // 释放显存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    // 释放内存
    free(h_A);
    free(h_B);
    free(h_C);
    // 释放设备
    hipDeviceReset();
}

int main()
{
    matrix_multiplication_cublas(2, 2);
    return 0;
}